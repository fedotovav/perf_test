#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <chrono>

#include <hip/hip_runtime.h>

#include "../test.h"

using namespace std;

__global__ void va_warm_up_kernel( double * a )
{
   *a = 0;
}

void va_warm_up()
{
   double * warm_tmp;
   
   hipMalloc((void **) &warm_tmp, sizeof(double));
   
   va_warm_up_kernel<<< dim3(1), dim3(1) >>>(warm_tmp);
   
   hipFree(warm_tmp);
}

__global__ void vec_add_kernel( double * a, double * b, double * c, int size )
{
   int bx = blockIdx.x;
   int tx = threadIdx.x;
   
   int global_idx = bx * blockDim.x + tx;
   
   c[global_idx] = a[global_idx] + b[global_idx];
}

__global__ void vec_add_with_check_kernel( double * a, double * b, double * c, int size )
{
   int bx = blockIdx.x;
   int tx = threadIdx.x;
   
   int global_idx = bx * blockDim.x + tx;
   
   if (global_idx < size)
      c[global_idx] = a[global_idx] + b[global_idx];
}

int va_device_mem_alloc( double ** dev_a, double ** dev_b, double ** dev_c, const double *& host_a, const double *& host_b, size_t size )
{
   hipError_t error = hipMalloc((void **)dev_a, sizeof(double) * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
      return 0;
   }

   error = hipMalloc((void **)dev_b, sizeof(double) * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
      return 0;
   }

   error = hipMalloc((void **)dev_c, sizeof(double) * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
      return 0;
   }

   error = hipMemcpy(*dev_a, host_a, sizeof(double) * size, hipMemcpyHostToDevice);

   if (error != hipSuccess)
   {
      printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
      return 0;
   }

   error = hipMemcpy(*dev_b, host_b, sizeof(double) * size, hipMemcpyHostToDevice);

   if (error != hipSuccess)
   {
      printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
      return 0;
   }
   
   return 1;
}

time_res_t vec_add( const double * a, const double * b, double * c, int block_size, size_t size )
{
   double   * d_A = NULL
          , * d_B = NULL
          , * d_C = NULL;

   hipError_t error;
   
   time_res_t time_res;
   
   va_warm_up();
   
   time_res.measure_start();
   
   va_device_mem_alloc(&d_A, &d_B, &d_C, a, b, size);

   time_res.mem_allocate_time_ = time_res.measure_finish();

   dim3 threads, grid;

   if (block_size < size)
   {
      threads.x = block_size;
      grid.x    = size / threads.x + 1;
   }
   else
   {
      threads.x = size;
      grid.x    = 1;
   }

   time_res.measure_start();

   vec_add_kernel<<< grid, threads >>>(d_A, d_B, d_C, size);

   hipDeviceSynchronize();

   time_res.computing_time_ = time_res.measure_finish();

   time_res.measure_start();

   error = hipMemcpy(c, d_C, sizeof(double) * size, hipMemcpyDeviceToHost);

   time_res.mem_allocate_time_ += time_res.measure_finish();

   if (error != hipSuccess)
   {
      printf("hipMemcpy(h_C, d_C) returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   hipDeviceReset();

   return time_res;
}

time_res_t vec_add_with_check( const double * a, const double * b, double * c, int block_size, int size )
 {
   double   * d_A = NULL
          , * d_B = NULL
          , * d_C = NULL;

   hipError_t error;
   
   time_res_t time_res;
   
   va_warm_up();
   
   time_res.measure_start();
   
   va_device_mem_alloc(&d_A, &d_B, &d_C, a, b, size);

   time_res.mem_allocate_time_ = time_res.measure_finish();

   dim3 threads, grid;

   if (block_size < size)
   {
      threads.x = block_size;
      grid.x    = size / threads.x + 1;
   }
   else
   {
      threads.x = size;
      grid.x    = 1;
   }

   time_res.measure_start();

   vec_add_with_check_kernel<<< grid, threads >>>(d_A, d_B, d_C, size);

   hipDeviceSynchronize();

   time_res.computing_time_ = time_res.measure_finish();

   time_res.measure_start();

   // Copy result from device to host
   error = hipMemcpy(c, d_C, sizeof(double) * size, hipMemcpyDeviceToHost);

   time_res.mem_allocate_time_ += time_res.measure_finish();

   if (error != hipSuccess)
   {
      printf("hipMemcpy (h_C, d_C) returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   hipDeviceReset();

   return time_res;
}

time_res_t va_calc_cu( int size, const double * a, const double * b, double * c )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return time_res_t();
   }

   if (error != hipSuccess)
   {
      printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
   }

   // Use a larger block size for Fermi and above
   int block_size = (deviceProp.major < 2) ? 16 : 32;

   time_res_t duration = vec_add(a, b, c, block_size, size);

   return duration;
}

time_res_t va_calc_cu_with_check( int size, const double * a, const double * b, double * c )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return time_res_t();
   }

   if (error != hipSuccess)
   {
      printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
   }

   // Use a larger block size for Fermi and above
   int block_size = (deviceProp.major < 2) ? 16 : 32;

   time_res_t duration = vec_add_with_check(a, b, c, block_size, size);

   return duration;
}
