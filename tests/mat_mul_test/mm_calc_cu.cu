#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>

#include <chrono>

#include <hip/hip_runtime.h>

#include "../test.h"

using namespace std;

// Compute C = A * B

template <int BLOCK_SIZE>
__global__ void matrixMulCUDA(double *C, double *A, double *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    double Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
time_res_t matrixMultiply(const double * a, const double * b, double * c, int block_size, int size)
{
   // Allocate device memory
   double *d_A, *d_B, *d_C;

   hipError_t error;
   
   time_res_t time_res;
   
   chrono::time_point<chrono::system_clock> time_start, time_finish;

   time_start = chrono::system_clock::now();

   error = hipMalloc((void **) &d_A, sizeof(double) * size * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   error = hipMalloc((void **) &d_B, sizeof(double) * size * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   error = hipMalloc((void **) &d_C, sizeof(double) * size * size);

   if (error != hipSuccess)
   {
      printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   // copy host memory to device
   error = hipMemcpy(d_A, a, sizeof(double) * size * size, hipMemcpyHostToDevice);

   if (error != hipSuccess)
   {
      printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   error = hipMemcpy(d_B, b, sizeof(double) * size * size, hipMemcpyHostToDevice);

   if (error != hipSuccess)
   {
      printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   time_finish = chrono::system_clock::now();

   size_t duration = chrono::duration_cast<std::chrono::milliseconds>(time_finish - time_start).count();
   
   time_res.mem_allocate_time_ = duration;

   // Setup execution parameters
   dim3 threads(block_size, block_size);
   dim3 grid(size / threads.x + 1, size / threads.y + 1);

   time_start = chrono::system_clock::now();

   // Performs warmup operation using matrixMul CUDA kernel
   if (block_size == 16)
   {
      matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, size, size);
   }
   else
   {
      matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, size, size);
   }

   hipDeviceSynchronize();

   time_finish = chrono::system_clock::now();

   duration = chrono::duration_cast<std::chrono::milliseconds>(time_finish - time_start).count();
   
   time_res.computing_time_ = duration;

   time_start = chrono::system_clock::now();

   // Copy result from device to host
   error = hipMemcpy(c, d_C, sizeof(double) * size * size, hipMemcpyDeviceToHost);

   time_finish = chrono::system_clock::now();

   duration = chrono::duration_cast<std::chrono::milliseconds>(time_finish - time_start).count();
   
   time_res.mem_allocate_time_ = duration;

   if (error != hipSuccess)
   {
      printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
      return time_res_t();
   }

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   hipDeviceReset();

   return time_res;
}

time_res_t mm_calc_cu( int size, const double * a, const double * b, double * c )
{
   int devID = 0;

   hipError_t error;
   hipDeviceProp_t deviceProp;
   error = hipGetDevice(&devID);

   if (error != hipSuccess)
   {
      printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
   }

   error = hipGetDeviceProperties(&deviceProp, devID);

   if (deviceProp.computeMode == hipComputeModeProhibited)
   {
      fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
      return time_res_t();
   }

   if (error != hipSuccess)
   {
      printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
   }

   // Use a larger block size for Fermi and above
   int block_size = (deviceProp.major < 2) ? 16 : 32;

   time_res_t duration = matrixMultiply(a, b, c, block_size, size);

   return duration;
}
